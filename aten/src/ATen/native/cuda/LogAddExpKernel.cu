#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/BinaryOps.h>
#include <ATen/AccumulateType.h>
#include <c10/util/MathConstants.h>

// NOTE: CUDA on Windows requires that the enclosing function
// of a __device__ lambda not have internal linkage.

namespace at { namespace native {

void logaddexp_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      ScalarType::BFloat16, ScalarType::Half,
      iter.dtype(), "logaddexp_cuda",
      [&]() {
        using accscalar_t = at::acc_type<scalar_t, /*is_cuda=*/true>;
        gpu_kernel(iter, [] GPU_LAMBDA (scalar_t a, scalar_t b) -> scalar_t {
          if (::isinf(static_cast<accscalar_t>(a)) && a == b) {
            return a;
          }
          else {
            scalar_t m = ::max(a, b);
            return m + ::log1p(::exp(-::abs(a - b)));
          }
        });
      });
}

void logaddexp2_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND(
      ScalarType::BFloat16,
      iter.dtype(), "logaddexp2_cuda",
      [&]() {
        using accscalar_t = at::acc_type<scalar_t, /*is_cuda=*/true>;
        const auto inv_log_2 = static_cast<accscalar_t>(1.0 / c10::ln_2<double>);
        gpu_kernel(iter, [inv_log_2] GPU_LAMBDA (scalar_t a, scalar_t b) -> scalar_t {
          if (::isinf(static_cast<accscalar_t>(a)) && a == b) {
            return a;
          }
          else {
            scalar_t m = ::max(a, b);
            return m + ::log1p(::exp2(-::abs(a - b))) * inv_log_2;
          }
        });
      });
}

REGISTER_DISPATCH(logaddexp_stub, &logaddexp_kernel_cuda);
REGISTER_DISPATCH(logaddexp2_stub, &logaddexp2_kernel_cuda);

}} // namespace at::native
